#include "hip/hip_runtime.h"
#include "mex.h"
#include "matrix.h"
#include "gpu/mxGPUArray.h"

void __global__ compute_deriv(double const * const T,
						double const * const P,
						double * const delta,
						int const nDim, int const nExamples)
{
	int const x = blockDim.x * blockIdx.x + threadIdx.x;
	if(x >= nDim*nExamples) return;

	unsigned int m = x/nDim;
	unsigned int j = x%nDim;

	unsigned int k,l;
	double delta_sum = 0.0;
	unsigned int n_pos_labels = 0;

	if(T[j+m*nDim]== 1) {			// if label i for instance j is positive
		for(l=0;l<nDim;l++) if(T[l+m*nDim]==-1) delta_sum += exp(-P[j+m*nDim]+P[l+m*nDim]);
		delta_sum = -delta_sum;
	} else if(T[j+m*nDim]==-1) {	// if label l is negative
		for(k=0;k<nDim;k++) if(T[k+m*nDim]== 1) delta_sum += exp(-P[k+m*nDim]+P[j+m*nDim]);
	}
	for(k=0;k<nDim;k++) if(T[k+m*nDim]==1) n_pos_labels++;
	delta[j+m*nDim] = 1.0/(n_pos_labels*(nDim-n_pos_labels))*delta_sum;
}

void mexFunction(int nlhs, mxArray *plhs[], /* output variables */
				int nrhs, const mxArray *prhs[])
{
	mxGPUArray const *T;
	mxGPUArray const *P;
	mxGPUArray *delta;
	double const *d_T, *d_P;
	double *d_delta;

	/* Choose a reasonably sized number of threads for the block. */
	int const threadsPerBlock = 256;
	int blocksPerGrid;
	int N;

	/* Initialize the GPU API */
	mxInitGPU();

	if ((nrhs!=2) || !(mxIsGPUArray(prhs[0])) || !(mxIsGPUArray(prhs[1]))) {
		mexErrMsgIdAndTxt("parallel:gpu:BPMLL:InvalidInput", "Invalid input to MEX file.");
	}

	T = mxGPUCreateFromMxArray(prhs[0]);	/* targets */
	P = mxGPUCreateFromMxArray(prhs[1]);	/* predictions */

	/* Verify that P and T really are double arrays before extracting the pointer. */
	if ((mxGPUGetClassID(T) != mxDOUBLE_CLASS) || (mxGPUGetClassID(P) != mxDOUBLE_CLASS)) {
		mexErrMsgIdAndTxt("parallel:gpu:BPMLL:InvalidInput", "Invalid input to MEX file.");
	}

	d_T = (double const *)(mxGPUGetDataReadOnly(T));
	d_P = (double const *)(mxGPUGetDataReadOnly(P));

	mwSize const *delta_dims = mxGPUGetDimensions(P);
	int const nDim = delta_dims[0];
	int const nExamples = delta_dims[2];

	/* Create a GPUArray to hold the result and get its underlying pointer. */

	delta = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(P),
								delta_dims,
								mxGPUGetClassID(P),
								mxGPUGetComplexity(P),
								MX_GPU_DO_NOT_INITIALIZE);
	d_delta = (double *)(mxGPUGetData(delta));

	N = (int)(mxGPUGetNumberOfElements(P));
	blocksPerGrid = (N+threadsPerBlock-1)/threadsPerBlock;
	compute_deriv<<<blocksPerGrid, threadsPerBlock>>>(d_T,d_P,d_delta,nDim,nExamples);

	/* Wrap the result up as a MATLAB gpuArray for return. */
	plhs[0] = mxGPUCreateMxArrayOnGPU(delta);

	mxGPUDestroyGPUArray(T);
	mxGPUDestroyGPUArray(P);
	mxGPUDestroyGPUArray(delta);

	mxFree((void*)delta_dims);

	return;
}
